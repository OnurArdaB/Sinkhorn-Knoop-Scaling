#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <omp.h>
#include <math.h>       /* fabsf */
#include <string.h>
#include <stdlib.h>
#include <time.h>

#define DEBUG 0

//Error check-----
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}
//Error check-----
//This is a very good idea to wrap your calls with that function.. Otherwise you will not be able to see what is the error.
//Moreover, you may also want to look at how to use cuda-memcheck and cuda-gdb for debugging.

__global__ void scalesk(){
  
  //TO DO: GPU SCALE
  
}

void wrapper(int* adj, int* xadj, int* tadj, int* txadj, double* rv, double* cv, int* nov, int* nnz, int siter){
  
  printf("Wrapper here! \n");
  
  //TO DO: DRIVER CODE
  
  hipEventCreate(&start);
  hipEventRecord(start, 0);
  
  scalesk<<<NO_BLOCKS,NO_THREADS>>>();
  gpuErrchk( hipDeviceSynchronize() );
  
  
  hipEventCreate(&stop);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("GPU scale took: %f s\n", elapsedTime/1000);
  
    
}

